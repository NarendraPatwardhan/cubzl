#include "hip/hip_runtime.h"
#include "kernel.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int rows_A, int cols_A, int cols_B) {
    // Compute the indices for the current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows_A && col < cols_B) {
        float sum = 0.0f;
        for (int k = 0; k < cols_A; ++k) {
            sum += A[row * cols_A + k] * B[k * cols_B + col];
        }
        C[row * cols_B + col] = sum;
    }
}

void matrixMultiplicationGPU(float* A, float* B, float* C, int rows_A, int cols_A, int cols_B) {
    // Allocate memory on the GPU
    float* d_A, *d_B, *d_C;
    size_t size_A = rows_A * cols_A * sizeof(float);
    size_t size_B = cols_A * cols_B * sizeof(float);
    size_t size_C = rows_A * cols_B * sizeof(float);

    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    // Copy data from the host (CPU) to the device (GPU)
    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    // Define grid and block dimensions for the kernel launch
    dim3 blockDim(32, 32);
    dim3 gridDim((cols_B + blockDim.x - 1) / blockDim.x, (rows_A + blockDim.y - 1) / blockDim.y);

    // Launch the CUDA kernel
    matrixMultiplicationKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, rows_A, cols_A, cols_B);

    // Copy the result back from the device (GPU) to the host (CPU)
    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    // Free memory on the GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}